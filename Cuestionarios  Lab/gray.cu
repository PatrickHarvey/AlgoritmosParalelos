#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <string>
#include <fstream>

using namespace std;

#define T 2 
#define N 512

#define CHANNELS 3

__global__
void colorToGray(const char * rgbImage, unsigned char * grayImage, int ancho, int altura)
{
    int columna = threadIdx.x + blockIdx.x * blockDim.x;
    int fila    = threadIdx.y + blockIdx.y * blockDim.y;

    if (columna < ancho && fila < altura)
     {

      int greyOffset = fila*(ancho) + columna;
      int rgbOffset = greyOffset * CHANNELS;
        unsigned char r = rgbImage[rgbOffset];
        unsigned char g = rgbImage[rgbOffset + 2];
        unsigned char b = rgbImage[rgbOffset + 3];
      grayImage[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}

int main(int argc, char** argv) {

  int entrada[512][512*3];
  int salida[512][512];

  unsigned char *m_entrada, *m_salida;
  
  const char * archivo_entrada="lena30.jpg";

  hipMalloc((void**) &m_entrada, N * CHANNELS * N * sizeof(int));
  hipMalloc((void**) &m_salida, N * N * sizeof(int));

  hipMemcpy(m_entrada, entrada, N * CHANNELS * N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(m_salida, salida, N * N * sizeof(int), hipMemcpyHostToDevice);

  int B = (int) ceil((float) N / (float) T);

  dim3 dimBloques(B, B);
  dim3 dimThreadsBloque(T, T);

  int ancho=N;
  int altura=N;
  colorToGray<<<dimBloques, dimThreadsBloque>>>(archivo_entrada, m_salida, ancho, altura);

  hipMemcpy(salida, m_salida, N * N * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(m_entrada);
  hipFree(m_salida);

 ofstream ficheroSalida;
 ficheroSalida.open ("gris.jpg");
 ficheroSalida << m_salida;
 ficheroSalida.close();
 
 return 0;
}
