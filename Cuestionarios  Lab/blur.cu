#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <fstream>

using namespace std;

#define T 2 // max threads x bloque

#define N 512

#define BLUR_SIZE 1


__global__
   void blurKernel(const char * in, unsigned char * out, int ancho, int altura) {
     int columna = blockIdx.x * blockDim.x + threadIdx.x;
     int fila = blockIdx.y * blockDim.y + threadIdx.y;
     if (columna < ancho && fila < altura) {
       int pixVal = 0;
       int pixels = 0;
       for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
         for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {
           int curRow = fila + blurRow;
           int curCol = columna + blurCol;
           if(curRow > -1 && curRow < altura && curCol > -1 && curCol < ancho){
             pixVal += in[curRow * ancho + curCol];
             pixels++;
           }
         }
       }
       out[fila * ancho + columna] = (int)(pixVal / pixels);
    }
 }


int main(int argc, char** argv) {

  int entrada[512][512*3];
  int salida[512][512];

  unsigned char *m_entrada, *m_salida;

  const char * archivo_entrada="lena30.jpg";

  hipMalloc((void**) m_entrada, N * N * 3 *sizeof(int));
  hipMalloc((void**) m_salida, N * N * sizeof(int));

  hipMemcpy(m_entrada, entrada, N * N * 3 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(m_salida, salida, N * N * sizeof(int), hipMemcpyHostToDevice);

  int B = (int) ceil((float) N / (float) T);

  dim3 dimBloques(B, B);
  dim3 dimThreadsBloque(T, T);

  int ancho=N;
  int altura=N;
  blurKernel<<<dimBloques, dimThreadsBloque>>>(archivo_entrada, m_salida, ancho, altura);

  hipMemcpy(salida, m_salida, N * N * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(m_entrada);
  hipFree(m_salida);

 ofstream ficheroSalida;
 ficheroSalida.open ("blur.jpg");
 ficheroSalida << m_salida;
 ficheroSalida.close();

        return 0;
}
