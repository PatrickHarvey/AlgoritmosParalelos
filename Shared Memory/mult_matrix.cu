#include "hip/hip_runtime.h"
#define TILE_WIDTH 16
#include <iostream>

using namespace std;

__global__ void matrixMultiply(float * A, float * B, float * C, int tam)
{
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty, Col = bx * TILE_WIDTH + tx;
    float Pvalue = 0;

    for (int ph = 0; ph < tam/TILE_WIDTH; ++ph)
    {
	ds_A[ty][tx] = A[Row*tam + ph*TILE_WIDTH + tx];

	ds_B[ty][tx] = B[(ph*TILE_WIDTH + ty)*tam + Col];

	__syncthreads();

	for (int k = 0; k < TILE_WIDTH; ++k)
            {
	    Pvalue += ds_A[ty][k] * ds_B[k][tx];
	    }

	__syncthreads();
     }
C[Row*tam + Col] = Pvalue;
}

int main(int argc, char ** argv) {

    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;

    int tam=16;

    hostA = (float *)malloc(sizeof(float) * tam * tam);
    hostB = (float *)malloc(sizeof(float) * tam * tam);
    hostC = (float *)malloc(sizeof(float) * tam * tam);

    

    for(int i=0;i<tam;i++)
	{
	hostA[i]=i;
	hostB[i]=tam-i;
	}

for(int j=0;j<tam;j++)
	cout << hostA[j] << " ";
cout << endl;
for(int k=0;k<tam;k++)
	cout << hostB[k] << " ";
cout << endl;
    hipMalloc(&deviceA, sizeof(float) * tam * tam);
    hipMalloc(&deviceB, sizeof(float) * tam * tam);
    hipMalloc(&deviceC, sizeof(float) * tam * tam);

    hipMemcpy(deviceA, hostA, sizeof(float) * tam * tam, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(float) * tam * tam, hipMemcpyHostToDevice);

    dim3 dimGrid((tam-1)/TILE_WIDTH+1, (tam-1)/TILE_WIDTH+1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, tam);

    hipDeviceSynchronize();

    hipMemcpy(hostC, deviceC, sizeof(float) * tam * tam, hipMemcpyDeviceToHost);

    for(int k=0;k<tam;k++)
	cout << hostC[k] << " ";
    cout << endl;

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);


    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
