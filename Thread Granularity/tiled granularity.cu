#define N 512
#define TILE_WIDTH 16

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;


__global__ void matrix_mult(int* A, int* B, int* C, int ancho)
{
int tmp = 0;
 int columna = blockIdx.x*TILE_WIDTH + threadIdx.x;
 int fila = blockIdx.y*TILE_WIDTH + threadIdx.y;
 if(columna < ancho && fila < ancho) {
  for (int k = 0; k < ancho; k++)
   tmp += A[fila * ancho + k] * B[k * ancho + columna];
  C[fila * ancho + columna] = tmp;
 }
}

int main() {
 int a[N][N], b[N][N], c[N][N];

 int *dev_a, *dev_b, *dev_c;

 int size = N * N * sizeof(int);

 // initialize a and b matrices here
 hipMalloc((void **) &dev_a, size);
 hipMalloc((void **) &dev_b, size);
 hipMalloc((void **) &dev_c, size);

 hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

 dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
 dim3 dimGrid((int)ceil(N/dimBlock.x), (int)ceil(N/dimBlock.y));

 matrix_mult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);

 hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

 hipFree(dev_a);
 hipFree(dev_b);
 hipFree(dev_c);
}

